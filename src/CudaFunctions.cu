#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Structures.h"
#include "Constantes.h"
#include "CudaFunctions.h"

/*
__global__
void multiplica(Matrix* matrixA, Matrix* matrixB, Matrix* matrixRes,float *test )
{
  int i;
  int j;

  MatrixType value;

  //Determino la posición de la matriz según bloque y thread

  //printf("Valor size i %zd %zd\n",matrixRes->size.i,matrixRes->size.j);

  i =  threadIdx.x;
  j =  threadIdx.y;

  i=0;
  j=0;

  value = -4.0f;

  int index = i*matrixRes->size.j+j;
  matrixRes->matrixValues[index] = value;

  *test = value;

}
*/

/*
Matrix* moveMatrix_2_device(Matrix *matrix)
{
  Matrix* localMatrix;
  Matrix* p_return = NULL;
  MatrixType* data;


  localMatrix = (Matrix*)malloc(sizeof(Matrix));
  if (localMatrix == NULL){
    return p_return;
  }


  hipMalloc((void **)data,matrix->size.i*matrix->size.j*sizeof(MatrixType));
  hipMemcpy((void *)data,(void *)matrix->matrixValues,matrix->size.i*matrix->size.j*sizeof(MatrixType),hipMemcpyHostToDevice);

  localMatrix->matrixValues = data;
  localMatrix->size.i = matrix->size.i;
  localMatrix->size.j = matrix->size.j;

  hipMalloc((void **)&p_return,sizeof(Matrix));
  hipMemcpy(p_return,localMatrix,sizeof(Matrix),hipMemcpyHostToDevice);

  return p_return;

}
*/
