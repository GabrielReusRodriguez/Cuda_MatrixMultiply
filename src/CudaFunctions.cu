#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Structures.h"
#include "Constantes.h"
#include "CudaFunctions.h"


__host__ __device__
MyDim3 newMyDim3(int x, int y, int z)
{
  MyDim3 dim;
    dim.x = x;
    dim.y = y;
    dim.z = z;
  return dim;

}

__global__
void cuda_multiplica(Matrix* matrixA, Matrix* matrixB, Matrix* matrixRes)
{
  int x;
  int y;
  int z;

  MatrixType value;

  //Determino la posición de la matriz según bloque y thread
  //x =  threadIdx.x;
  //y =  threadIdx.y;

   MyDim3 blockGrid = newMyDim3(blockIdx.x,blockIdx.y,blockIdx.z);
   MyDim3 threadId  = newMyDim3(threadIdx.x,threadIdx.y,threadIdx.z);

   MyDim3 myBlockDim = newMyDim3(blockDim.x,blockDim.y,blockDim.z);
   MyDim3 myGridDim  = newMyDim3(gridDim.x,gridDim.y,gridDim.z);

  calculaCoordenadasMatriz(blockGrid,threadId,x,y,z);

  value = 0.0f;

  int index = y*matrixRes->size.i+x;

  for(int desp = 0;desp < matrixA->size.i;desp++ )
  {
    int ii_A=desp;
    int jj_A=y;
    int ii_B=x;
    int jj_B=desp;

    int index_A = jj_A*matrixA->size.i+ii_A;
    int index_B = jj_B*matrixB->size.i+ii_B;
    value+= matrixA->matrixValues[index_A]*matrixB->matrixValues[index_B];
  }

  matrixRes->matrixValues[index] = value;

}


void multiplica(Matrix* d_matrix_A, Matrix* d_matrix_B, Matrix* d_matrix_Res,MyDim3 p_dimBlock, MyDim3 p_dimGrid)
{

  int x,y,z;

  x =  p_dimBlock.x;
  y =  p_dimBlock.y;

  if(p_dimBlock.z == 0){
      z = 1;
  }else{
      z =  p_dimBlock.z;
  }

  dim3 dimBlock(x,y,z);

  x =  p_dimGrid.x;
  y =  p_dimGrid.y;

  if(p_dimGrid.z == 0){
      z = 1;
  }else{
      z =  p_dimGrid.z;
  }

  dim3 dimGrid(x,y,z);

  cuda_multiplica<<<dimGrid, dimBlock>>>(d_matrix_A, d_matrix_B, d_matrix_Res);

}

Matrix* moveMatrix_2_device(Matrix *h_matrix)
{
  Matrix* localMatrix;
  Matrix* p_return = NULL;
  MatrixType* d_data;

  localMatrix = (Matrix*)malloc(sizeof(Matrix));
  if (localMatrix == NULL){
    return NULL;
  }

  const size_t data_size = sizeof(MatrixType) * size_t(h_matrix->size.i*h_matrix->size.j);

  hipMalloc((void **)&d_data,data_size);
  hipMemcpy((void *)d_data, (void *) h_matrix->matrixValues ,data_size, hipMemcpyHostToDevice);
  localMatrix->matrixValues = d_data;
  localMatrix->size.i = h_matrix->size.i;
  localMatrix->size.j = h_matrix->size.j;

  hipMalloc((void **)&p_return,sizeof(Matrix));
  hipMemcpy(p_return,localMatrix,sizeof(Matrix),hipMemcpyHostToDevice);

  //Liberamos la memoria de la estructura Matrix.
  free(localMatrix);

  return p_return;

}


Matrix* moveMatrix_2_host(Matrix *d_matrix, int size_i, int size_j)
{
  Matrix* localMatrix;

  localMatrix = (Matrix*)malloc(sizeof(Matrix));
  if (localMatrix == NULL){
    return NULL;
  }

  const size_t data_size = sizeof(MatrixType) * size_t(size_i*size_j);

  localMatrix->matrixValues = (MatrixType*)malloc(data_size);

  MatrixType* d_data;
  MatrixType* h_data;

  hipMemcpy((void *)localMatrix,(void *)d_matrix,sizeof(Matrix),hipMemcpyDeviceToHost);

  d_data = localMatrix->matrixValues;
  h_data = (MatrixType *)malloc(data_size);
  hipMemcpy((void *)h_data,(void *)d_data,data_size,hipMemcpyDeviceToHost);
  localMatrix->matrixValues = h_data;

  return localMatrix;

}


hipDeviceProp_t getCudaProperties()
{
  int nDevices;
  hipDeviceProp_t prop;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipGetDeviceProperties(&prop, i);

   printf("Device Number: %d\n", i);
   printf("  Device name: %s\n", prop.name);
   printf("  Memory Clock Rate (KHz): %d\n",
          prop.memoryClockRate);
   printf("  Memory Bus Width (bits): %d\n",
          prop.memoryBusWidth);
   printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
          2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
   printf("Maximum threads per block: %d\n",prop.maxThreadsPerBlock);
   printf("Maximum blocks: x: %d y:%d z: %d\n",prop.maxGridSize[0],
          prop.maxGridSize[1],prop.maxGridSize[2]);

 }

 return prop;

}

__device__
void calculaCoordenadasMatriz(MyDim3 blockGrid, MyDim3 threadId, int &x, int &y, int &z)
{
    x = threadId.x;
    y = threadId.y;
    z = threadId.z;
}
