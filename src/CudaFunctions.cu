#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Structures.h"
#include "Constantes.h"
#include "CudaFunctions.h"



MyDim3 newMyDim3(int x, int y, int z)
{
  MyDim3 dim;
    dim.x = x;
    dim.y = y;
    dim.z = z;
  return dim;

}

__global__
void cuda_multiplica(Matrix* matrixA, Matrix* matrixB, Matrix* matrixRes)
{
  int i;
  int j;

  MatrixType value;

  //Determino la posición de la matriz según bloque y thread
  i =  threadIdx.x;
  j =  threadIdx.y;

  value = 0.0f;

  int index = j*matrixRes->size.i+i;

  for(int desp = 0;desp < matrixA->size.i;desp++ )
  {
    int ii_A=desp;
    int jj_A=j;
    int ii_B=i;
    int jj_B=desp;

    int index_A = jj_A*matrixA->size.i+ii_A;
    int index_B = jj_B*matrixB->size.i+ii_B;
    value+= matrixA->matrixValues[index_A]*matrixB->matrixValues[index_B];
  }

  matrixRes->matrixValues[index] = value;

}


void multiplica(Matrix* d_matrix_A, Matrix* d_matrix_B, Matrix* d_matrix_Res,MyDim3 p_dimBlock, MyDim3 p_dimGrid)
{

  int x,y,z;

  x =  p_dimBlock.x;
  y =  p_dimBlock.y;

  if(p_dimBlock.z == 0){
      z = 1;
  }else{
      z =  p_dimBlock.z;
  }

  dim3 dimBlock(x,y,z);

  x =  p_dimGrid.x;
  y =  p_dimGrid.y;

  if(p_dimGrid.z == 0){
      z = 1;
  }else{
      z =  p_dimGrid.z;
  }

  dim3 dimGrid(x,y,z);

  cuda_multiplica<<<dimGrid, dimBlock>>>(d_matrix_A, d_matrix_B, d_matrix_Res);

}

Matrix* moveMatrix_2_device(Matrix *h_matrix)
{
  Matrix* localMatrix;
  Matrix* p_return = NULL;
  MatrixType* d_data;

  localMatrix = (Matrix*)malloc(sizeof(Matrix));
  if (localMatrix == NULL){
    return NULL;
  }

  const size_t data_size = sizeof(MatrixType) * size_t(h_matrix->size.i*h_matrix->size.j);

  hipMalloc((void **)&d_data,data_size);
  hipMemcpy((void *)d_data,(void *)h_matrix->matrixValues,data_size,hipMemcpyHostToDevice);

  localMatrix->matrixValues = d_data;
  localMatrix->size.i = h_matrix->size.i;
  localMatrix->size.j = h_matrix->size.j;

  hipMalloc((void **)&p_return,sizeof(Matrix));
  hipMemcpy(p_return,localMatrix,sizeof(Matrix),hipMemcpyHostToDevice);

  //Liberamos la memoria de la estructura Matrix.
  free(localMatrix);

  return p_return;

}


Matrix* moveMatrix_2_host(Matrix *d_matrix, int size_i, int size_j)
{
  Matrix* localMatrix;

  localMatrix = (Matrix*)malloc(sizeof(Matrix));
  if (localMatrix == NULL){
    return NULL;
  }

  const size_t data_size = sizeof(MatrixType) * size_t(size_i*size_j);

  localMatrix->matrixValues = (MatrixType*)malloc(data_size);

  MatrixType* d_data;
  MatrixType* h_data;

  hipMemcpy((void *)localMatrix,(void *)d_matrix,sizeof(Matrix),hipMemcpyDeviceToHost);

  d_data = localMatrix->matrixValues;
  h_data = (MatrixType *)malloc(data_size);
  hipMemcpy((void *)h_data,(void *)d_data,data_size,hipMemcpyDeviceToHost);
  localMatrix->matrixValues = h_data;

  return localMatrix;

}
