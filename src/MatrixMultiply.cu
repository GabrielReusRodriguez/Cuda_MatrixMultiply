#include<stdio.h>
#include "Structures.h"
#include "Constantes.h"
#include "MemoryLib.h"
#include "FileLib.h"
#include "MatrixLib.h"
#include "CudaFunctions.h"

int main(void){

	ConstString fileA = "./data/A.txt";
	ConstString fileB = "./data/B.txt";
  //ConstString fileA = "./data/3x3.txt";
  //ConstString fileB = "./data/Identity.txt";

	ConstString fileRes = "./data/Res.txt";

	Matrix h_matrizA;
	Matrix h_matrizB;
	Matrix h_matrizRes;

	MatrixSize matrixSize_Res;

	h_matrizA = leeFichero(fileA);
	if (h_matrizA.matrixValues == NULL){
		return 1;
	}
	h_matrizB = leeFichero(fileB);
	if (h_matrizB.matrixValues == NULL){
		return 1;
	}

	int returnValue = calculaTamanosMult(h_matrizA.size,h_matrizB.size,matrixSize_Res);
	if (returnValue != RETURN_OK){
		printf("Tamanos distintos\n");
		liberaMemoriaMatriz(h_matrizA);
		liberaMemoriaMatriz(h_matrizB);
		return 1;
	}

	h_matrizRes= reservaMemoriaMatriz(matrixSize_Res);
	if (h_matrizRes.matrixValues == NULL)
	{
		liberaMemoriaMatriz(h_matrizA);
		liberaMemoriaMatriz(h_matrizB);
		return 2;
	}

	//Empieza la multiplicacion.
	//TODO: https://www.cs.cmu.edu/afs/cs/academic/class/15668-s11/www/cuda-doc/html/group__CUDART__MEMORY_gb17fef862d4d1fefb9dba35bd62a187e.html

  Matrix* d_matrix_Res = moveMatrix_2_device(&h_matrizRes);
  Matrix* d_matrix_A = moveMatrix_2_device(&h_matrizA);
  Matrix* d_matrix_B = moveMatrix_2_device(&h_matrizB);

  if(d_matrix_Res == NULL || d_matrix_A == NULL || d_matrix_B == NULL){
    liberaMemoriaMatriz(h_matrizA);
    liberaMemoriaMatriz(h_matrizB);
    liberaMemoriaMatriz(h_matrizRes);
    return 3;
  }

  MyDim3 dimBlock = newMyDim3(matrixSize_Res.i,matrixSize_Res.j,1);
  MyDim3 dimGrid = newMyDim3(1,1,1);

  multiplica(d_matrix_A, d_matrix_B, d_matrix_Res, dimBlock, dimGrid);

  hipDeviceSynchronize();

  Matrix* h_matriz_Res_v2;

  h_matriz_Res_v2 = moveMatrix_2_host(d_matrix_Res,h_matrizRes.size.i,h_matrizRes.size.j);

	printMatrix(*h_matriz_Res_v2);

  escribeFichero(fileRes, *h_matriz_Res_v2);

	hipFree(d_matrix_A);
	hipFree(d_matrix_B);
	hipFree(d_matrix_Res);
	//Finalizamos.

	liberaMemoriaMatriz(h_matrizA);
	liberaMemoriaMatriz(h_matrizB);
	liberaMemoriaMatriz(h_matrizRes);
  liberaMemoriaMatriz(*h_matriz_Res_v2);

  free (h_matriz_Res_v2);

	return RETURN_OK;
}
