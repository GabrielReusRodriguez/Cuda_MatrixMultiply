#include "hip/hip_runtime.h"
#include<stdio.h>
#include "Structures.h"
#include "Constantes.h"
#include "MemoryLib.h"
#include "FileLib.h"
#include "MatrixLib.h"
#include "CudaFunctions.h"



__global__
void multiplica(Matrix* matrixA, Matrix* matrixB, Matrix* matrixRes,float *test )
{
  int i;
  int j;

  MatrixType value;

  //Determino la posición de la matriz según bloque y thread

  //printf("Valor size i %zd %zd\n",matrixRes->size.i,matrixRes->size.j);

  i =  threadIdx.x;
  j =  threadIdx.y;
/*
  i=0;
  j=0;
  */

  value = 0.0f;

  int index = j*matrixRes->size.i+i;

  for(int desp = 0;desp < matrixA->size.i;desp++ )
  {
    int ii_A=desp;
    int jj_A=j;
    int ii_B=i;
    int jj_B=desp;

    int index_A = jj_A*matrixA->size.i+ii_A;
    int index_B = jj_B*matrixB->size.i+ii_B;


    value+= matrixA->matrixValues[index_A]*matrixB->matrixValues[index_B];

    //value+= matrixA->matrixValues[index_A];
  }

  //value  = matrixA->matrixValues[index]+matrixB->matrixValues[index];

  matrixRes->matrixValues[index] = value;


//*test = (*matrixRes).matrixValues[1];
*test = matrixRes->matrixValues[index];

//*test = matrixRes->matrixValues[0];
//*test = value;
*test = matrixA->size.i;



  /*
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
  */
}


Matrix* moveMatrix_2_device(Matrix *h_matrix)
{
  Matrix* localMatrix;
  Matrix* p_return = NULL;
  MatrixType* d_data;

  localMatrix = (Matrix*)malloc(sizeof(Matrix));
  if (localMatrix == NULL){
    return p_return;
  }
  /*
  printf("pre memcpy\n");
  h_data = (MatrixType *)malloc(matrix->size.i*matrix->size.j*sizeof(MatrixType));
  memcpy(h_data,(*matrix).matrixValues,matrix->size.i*matrix->size.j*sizeof(MatrixType));
  printf("post memcpy\n");
*/

  hipMalloc((void **)&d_data,h_matrix->size.i*h_matrix->size.j*sizeof(MatrixType));
  hipMemcpy((void *)d_data,(void *)h_matrix->matrixValues,h_matrix->size.i*h_matrix->size.j*sizeof(MatrixType),hipMemcpyHostToDevice);
  //hipMemcpy((void *)d_data,h_data,matrix->size.i*matrix->size.j*sizeof(MatrixType),hipMemcpyHostToDevice);

  localMatrix->matrixValues = d_data;
  localMatrix->size.i = h_matrix->size.i;
  localMatrix->size.j = h_matrix->size.j;

  hipMalloc((void **)&p_return,sizeof(Matrix));
  hipMemcpy(p_return,localMatrix,sizeof(Matrix),hipMemcpyHostToDevice);

  return p_return;

}


Matrix* moveMatrix_2_host(Matrix *d_matrix, int size_i, int size_j)
{
  Matrix* localMatrix;
  Matrix* p_return = NULL;


  localMatrix = (Matrix*)malloc(sizeof(Matrix));
  if (localMatrix == NULL){
    printf("NULLACO \n");
    return p_return;
  }

  localMatrix->matrixValues = (MatrixType*)malloc(size_i*size_j*sizeof(MatrixType));

  MatrixType* d_data;
  MatrixType* h_data;

  hipMemcpy((void *)localMatrix,(void *)d_matrix,sizeof(Matrix),hipMemcpyDeviceToHost);

  d_data = localMatrix->matrixValues;
  h_data = (MatrixType *)malloc(size_i*size_j*sizeof(MatrixType));
  hipMemcpy((void *)h_data,(void *)d_data,size_i*size_j*sizeof(MatrixType),hipMemcpyDeviceToHost);
  localMatrix->matrixValues = h_data;

  return localMatrix;

}



int main(void){

	//ConstString fileA = "./data/A.txt";
	//ConstString fileB = "./data/B.txt";
  ConstString fileA = "./data/3x3.txt";
  ConstString fileB = "./data/Identity.txt";

	ConstString fileRes = "./data/Res.txt";

	Matrix h_matrizA;
	Matrix h_matrizB;
	Matrix h_matrizRes;

	MatrixSize matrixSize_Res;


	h_matrizA = leeFichero(fileA);
	if (h_matrizA.matrixValues == NULL){
		return 1;
	}
	h_matrizB = leeFichero(fileB);
	if (h_matrizB.matrixValues == NULL){
		return 1;
	}

	int returnValue = calculaTamanosMult(h_matrizA.size,h_matrizB.size,matrixSize_Res);
	if (returnValue != RETURN_OK){
		printf("Tamanos distintos\n");
		liberaMemoriaMatriz(h_matrizA);
		liberaMemoriaMatriz(h_matrizB);
		return 1;
	}

	h_matrizRes= reservaMemoriaMatriz(matrixSize_Res);
	if (h_matrizRes.matrixValues == NULL)
	{
		liberaMemoriaMatriz(h_matrizA);
		liberaMemoriaMatriz(h_matrizB);
		return 2;
	}

	//Empieza la multiplicacion.
	//TODO: https://www.cs.cmu.edu/afs/cs/academic/class/15668-s11/www/cuda-doc/html/group__CUDART__MEMORY_gb17fef862d4d1fefb9dba35bd62a187e.html
	Matrix* d_matrix_A;
	const size_t a_size = sizeof(MatrixType) * size_t(h_matrizA.size.i*h_matrizA.size.j);

	Matrix* d_matrix_B;
	const size_t b_size = sizeof(MatrixType) * size_t(h_matrizB.size.i*h_matrizB.size.j);

	Matrix* d_matrix_Res;
	const size_t res_size = sizeof(MatrixType) * size_t(h_matrizRes.size.i*h_matrizRes.size.j);
	//printf("Pre-malloc\n");

//http://horacio9573.no-ip.org/cuda/group__CUDART__MEMORY_g17f3a55e8c9aef5f90b67cdf22851375.html#g17f3a55e8c9aef5f90b67cdf22851375
//http://horacio9573.no-ip.org/cuda/group__CUDART__TYPES_g18fa99055ee694244a270e4d5101e95b.html#gg18fa99055ee694244a270e4d5101e95b783338534304281650c6cb1363f5a00a
	//printf("Post-FOR\n");
	h_matrizRes.matrixValues[1] = 5.0f;
 	///hipMalloc((void**)&d_matrix_Res,res_size);
	//hipMemcpy((void*)d_matrix_Res,(void*)&h_matrizRes,sizeof(Matrix),hipMemcpyHostToDevice);
  d_matrix_Res = moveMatrix_2_device(&h_matrizRes);
  d_matrix_A = moveMatrix_2_device(&h_matrizA);
  d_matrix_B = moveMatrix_2_device(&h_matrizB);

  if(d_matrix_Res == NULL){
    printf("NULLACO \n");
  }

	float *p_test;
	float test = -1.0f;
	hipMalloc((void**)&p_test,sizeof(float));
	hipMemcpy((void*)p_test,(void*)&test,sizeof(float),hipMemcpyHostToDevice);

		dim3 dimBlock( matrixSize_Res.i, matrixSize_Res.j );
		dim3 dimGrid( 1, 1 );

	multiplica<<<dimGrid, dimBlock>>>(d_matrix_A, d_matrix_B, d_matrix_Res,p_test);

	hipMemcpy((void*)&test,(void*)p_test,sizeof(float),hipMemcpyDeviceToHost);

	//printf("TEST: %f\n",test);

  Matrix* h_matriz_Res_v2;

  h_matriz_Res_v2 = moveMatrix_2_host(d_matrix_Res,h_matrizRes.size.i,h_matrizRes.size.j);
  //printf("Salida: \n");

	//hipMemcpy((void *)h_matrizRes.matrixValues, (void *)&(d_matrix_Res->matrixValues), res_size, hipMemcpyDeviceToHost);
	//hipMemcpy((void *)&h_matrizRes, (void *)d_matrix_Res, sizeof(Matrix), hipMemcpyDeviceToHost);

/*
	hipMemcpy((void *)&h_matrizRes, (void *)d_matrix_Res, sizeof(Matrix), hipMemcpyDeviceToHost);
	hipMemcpy((void *)&(h_matrizRes->matrixValues), (void *)d_matrix_Res.matrixValues, res_size, hipMemcpyHostToDevice);
	*/
	/*
	for (int i=0;i<h_matrizRes.size.i;i++)
	{
			//hipMalloc((void **)&(d_matrix_Res->matrixValues[i]),h_matrizRes.size.j*sizeof(MatrixType*));
			hipMemcpy(&(h_matrizRes.matrixValues[i]),d_matrix_Res->matrixValues[i], h_matrizRes.size.j*sizeof(MatrixType) , hipMemcpyHostToDevice);
	}
*/

	//printf("My Tamano: %zd %zd \n",h_matriz_Res_v2->size.i,h_matriz_Res_v2->size.j);
	printMatrix(*h_matriz_Res_v2);

	hipFree(d_matrix_A);
	hipFree(d_matrix_B);
	hipFree(d_matrix_Res);
	//Finalizamos.

	liberaMemoriaMatriz(h_matrizA);
	liberaMemoriaMatriz(h_matrizB);
	liberaMemoriaMatriz(h_matrizRes);
  liberaMemoriaMatriz(*h_matriz_Res_v2);

  free (h_matriz_Res_v2);

	//printf("Hola mundo!\n");
	return RETURN_OK;
}
